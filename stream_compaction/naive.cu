#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernNaiveScanStep(int* odata, int* idata, int n, int offset) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;

            if (index >= offset) {
                odata[index] = idata[index] + idata[index - offset];
            }
            else {
                odata[index] = idata[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            int* dev_odata;
            int* dev_idata;
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, n * sizeof(int));

            hipMemset(dev_idata, 0, sizeof(int));
            hipMemcpy(dev_idata+1, idata, (n-1) * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            int numSteps = ilog2ceil(n);
            for (int d = 1; d <= numSteps; d++) {
                int offset = 1 << (d - 1);
                int block_size = 512;
                dim3 num_blocks = (n + block_size - 1) / block_size;
                kernNaiveScanStep <<<num_blocks, block_size >> > (dev_odata, dev_idata, n, offset);
                hipDeviceSynchronize();
                std::swap(dev_odata, dev_idata);  
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_idata);
        }
    }
}
