#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernNaiveScanStep(int* odata, int* idata, int n, int offset) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;

            if (index >= offset) {
                odata[index] = idata[index] + idata[index - offset];
            }
            else {
                odata[index] = idata[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            int* dev_odata;
            int* dev_idata;
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_idata, n * sizeof(int));

            hipMemcpy(dev_idata, 0, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_idata+1, idata, (n-1) * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            int numSteps = ilog2ceil(n);
            for (int d = 1; d <= numSteps; d++) {
                int offset = 1 << (d - 1);
                kernNaiveScanStep <<<1, std::min(n, 1024) >> > (dev_odata, dev_idata, n, offset);
                std::swap(dev_odata, dev_idata);  
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_idata);
        }
    }
}
